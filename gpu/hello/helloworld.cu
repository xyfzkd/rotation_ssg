#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>
__global__ void helloFromGPU(void)
{
        printf("Hello World from GPU!\n");
        
}

int main(void)
{
        // hello from cpu
        hipError_t cudaStatus;
            printf("Hello World from CPU!\n");

                helloFromGPU << <1, 10 >> > ();
                    hipDeviceReset();//重置CUDA设备释放程序占用的资源
                        system("pause");
                            return 0;
                            
}


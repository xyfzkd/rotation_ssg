#include "hip/hip_runtime.h"
#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""

#define GPU

#define pi 3.1415926535
#define LENGTH 100000 //signal sampling points


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
#ifdef TEST
    float Data[LENGTH] = { 1,2,3,4 };
    float fs = 1000000.000;//sampling frequency
    float f0 = 200000.00;// signal frequency
    for (int i = 0; i < LENGTH; i++)
    {
        Data[i] = 1.35*cos(2 * pi*f0*i / fs);//signal gen,
    }

    hipfftComplex *CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
    int i;
    for (i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    for (i = 0; i < LENGTH / 2; i++)
    {
        printf("i=%d\tf= %6.1fHz\tRealAmp=%3.1f\t", i, fs*i / LENGTH, CompData[i].x*2.0 / LENGTH);
        printf("ImagAmp=+%3.1fi", CompData[i].y*2.0 / LENGTH);
        printf("\n");
    }
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);
#endif
#ifdef GPU
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);
    /* https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1 */
    hipfftHandle planIn;
    hipfftComplex *comp_data;
    hipfftReal *real_data;

//    if (hipGetLastError() != hipSuccess){
//        fprintf(stderr, "Cuda error: Failed to allocate\n");
//        return;
//    }

    hipMalloc((void**)&real_data, sizeof(hipfftComplex)*N[0]*N[1]);
    hipMalloc((void**)&comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));


    hipMemcpy(comp_data, (hipfftComplex*) MULTIDIM_ARRAY(src2), sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);
    printf("nihoa\n");
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(real_data, (hipfftReal*) MULTIDIM_ARRAY(dest), sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyHostToDevice);

    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */

    /* https://docs.nvidia.com/cuda/cufft/index.html 3.9.3 */

    hipfftExecC2R(planIn, comp_data, real_data);

    hipMemcpy(MULTIDIM_ARRAY(dest),real_data, sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipFree(comp_data);
    hipFree(real_data);
#endif
}

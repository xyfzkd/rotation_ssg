#include "hip/hip_runtime.h"
#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""
#include "time.h"
#include "src/multidim_array.h"

#define PRINTCOMP

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**********************************************************************/
/* function for simulating data for iFFT
 * input of type MultidimArray<fComplex> *, simulate random
 * data, and this function should be integrated into class MultidimArray
 * there is macro RELION_ALIGNED_MALLOC, with data simulator initRandom or others.
 * However, it seems as if it's designed for common type.
 * I decide to fix the omission.
 *
 *         T* ptr=NULL;
 *         long int n;
 *         if (mode == "uniform")
 *             FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY_ptr(*this,n,ptr)
 *             *ptr = static_cast< T >(rnd_unif(op1, op2));
 *
 *  what if T is fComplex?
 */
/***********************************************************************/

void rand_comp(MultidimArray<fComplex>& s){
    fComplex* ptr=NULL;
    long int n;
    FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY_ptr(s,n,ptr){
        (*ptr).real = (float) (rnd_unif(0, 1));
        (*ptr).imag = (float) (rnd_unif(0, 1));
    }


#ifdef PRINTCOMP
    printf("this is image, stored in array(%d, %d, %d, %d)\n", NSIZE(s), ZSIZE(s), YSIZE(s), XSIZE(s));
    for (int i=0; i < 16; i++){
        printf("%3.1f %3.1f \n", s.data[i].real,  s.data[i].imag);
    }
#endif
}

/**********************************************************************/
/* function for testing the resulting differences
 *          sum( abs(re1-re2) / (abs(re1+re2)+eps) )
 *
 * macros for traversing the same size MultidimArray, similar to src/multidim_array.h:234
 */
/***********************************************************************/

#define DIFF_ptr(re1,re2,n,ptr1,ptr2) \
    for ((n)=0, (ptr1)=(re1).data, (ptr2)=(re2).data; (n)<NZYXSIZE(re1); ++(n), ++(ptr1), ++(ptr1))


float diff(MultidimArray<float>& re1, MultidimArray<float>& re2){
    if(NSIZE(re1)!=NSIZE(re2) || \
       ZSIZE(re1)!=ZSIZE(re2) || \
       YSIZE(re1)!=ZSIZE(re2) || \
       XSIZE(re1)!=XSIZE(re2)){
        printf("Unequal dimensions:\n  Array1: (%d, %d, %d, %d)\n  Array1: (%d, %d, %d, %d)\n",
                NSIZE(re1), ZSIZE(re1), YSIZE(re1), XSIZE(re1),
                NSIZE(re2), ZSIZE(re2), YSIZE(re2), XSIZE(re2));
        return 0;
    }
    float *ptr1=NULL, *ptr2=NULL;
    long int n;
    float diff = 0, eps=1e-8;
    DIFF_ptr(re1,re2,n,ptr1,ptr2){
        diff += abs(*ptr1 - *ptr2) / (abs(*ptr1 + *ptr2) + eps);
    }
    printf("Difference is %f\n", diff);
}

/*******************************************************/
/* CuFFT for 2D image c2r realization
 * param@ src: MultidimArray<fComplex>, using array part
 * param@ dest: MultidimArray<float>, using array part
 */
/*******************************************************/

void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
    /* http://www.orangeowlsolutions.com/archives/1173 arct
     * https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1
     * https://docs.nvidia.com/cuda/cufft/index.html 3.9.3
     * https://www.beechwood.eu/using-cufft/ time
     * */
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);

    rand_comp(src2);


    float elapsedTime = 0;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipfftComplex *host_comp_data, *device_comp_data;
    hipfftReal    *host_real_data, *device_real_data;

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */
    host_comp_data = (hipfftComplex*) MULTIDIM_ARRAY(src2);
    host_real_data = MULTIDIM_ARRAY(dest);

    gpuErrchk(hipMalloc((void**)&device_real_data, sizeof(hipfftReal)*N[0]*N[1]));
    gpuErrchk(hipMalloc((void**)&device_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1)));


    hipMemcpy(device_comp_data, host_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);

    hipfftHandle planIn;


    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);


    hipfftExecC2R(planIn, device_comp_data, device_real_data);


    hipMemcpy(host_real_data, device_real_data, sizeof(hipfftReal)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipfftDestroy(planIn);
    gpuErrchk(hipFree(device_comp_data));
    gpuErrchk(hipFree(device_real_data));

    diff(dest,dest);

    //GET CALCULATION TIME
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("CUFFT Calculation COMPLETED IN : % 5.3f ms \n",elapsedTime);
}

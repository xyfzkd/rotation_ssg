#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""
#include "time.h"
#include "src/multidim_array.h"

#define PRINTCOMP

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**********************************************************************/
/* function for simulate data for iFFT
 * input of type MultidimArray<fComplex> *, simulate random
 * data, and this function should be integrated into class MultidimArray
 * there is macro RELION_ALIGNED_MALLOC, with data simulator initRandom or others.
 * However, it seems as if it's designed for common type.
 * I decide to fix the omission.
 *
 *         T* ptr=NULL;
 *         long int n;
 *         if (mode == "uniform")
 *             FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY_ptr(*this,n,ptr)
 *             *ptr = static_cast< T >(rnd_unif(op1, op2));
 *
 *  what if T is fComplex?
 */
/***********************************************************************/

void rand_comp(MultidimArray<fComplex>& s){
    T* ptr=NULL;
    long int n;
    FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY_ptr(*s,n,ptr)
    (*ptr).real = static_cast< T >(rnd_unif(op1, op2));
    (*ptr).imag = static_cast< T >(rnd_unif(op1, op2));

#ifdef PRINTCOMP
    for (int i=0; i < 16; i++){
        printf("%3.1f %3.1f \n", *s.data[i].real,  *s.data[i].imag)
    }
#endif
}

/*******************************************************/
/* CuFFT for 2D image c2r realization
 * param@ src: MultidimArray<fComplex>, using array part
 * param@ dest: MultidimArray<float>, using array part
 */
/*******************************************************/

void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
    /* http://www.orangeowlsolutions.com/archives/1173 arct
     * https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1
     * https://docs.nvidia.com/cuda/cufft/index.html 3.9.3
     * https://www.beechwood.eu/using-cufft/ time
     * */
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);

    float elapsedTime = 0;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipfftComplex *host_comp_data, *device_comp_data;
    hipfftReal    *host_real_data, *device_real_data;

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */
    host_comp_data = (hipfftComplex*) MULTIDIM_ARRAY(src2);
    host_real_data = MULTIDIM_ARRAY(dest);

    gpuErrchk(hipMalloc((void**)&device_real_data, sizeof(hipfftReal)*N[0]*N[1]));
    gpuErrchk(hipMalloc((void**)&device_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1)));


    hipMemcpy(device_comp_data, host_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);

    hipfftHandle planIn;


    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);


    hipfftExecC2R(planIn, device_comp_data, device_real_data);

    hipMemcpy(host_real_data, device_real_data, sizeof(hipfftReal)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipfftDestroy(planIn);
    gpuErrchk(hipFree(device_comp_data));
    gpuErrchk(hipFree(device_real_data));

    //GET CALCULATION TIME
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("CUFFT Calculation COMPLETED IN : % 5.3f ms \n",elapsedTime);
}

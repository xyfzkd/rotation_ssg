#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include "src/jaz/new_ft.h"

#include "src/macros.h"
#include "src/fftw.h"
#include "src/args.h"
#include <string.h>
#include <math.h>

#define CPU


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{

    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);
#ifdef GPU
    /* https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1 */
    hipfftHandle planIn;
    hipfftComplex *comp_data;
    hipfftReal *real_data;

//    if (hipGetLastError() != hipSuccess){
//        fprintf(stderr, "Cuda error: Failed to allocate\n");
//        return;
//    }

    hipMalloc((void**)&real_data, sizeof(hipfftComplex)*N[0]*N[1]);
    hipMalloc((void**)&comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));


    hipMemcpy(comp_data, (hipfftComplex*) MULTIDIM_ARRAY(src2), sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);
    hipStreamSynchronize(0);
    hipMemcpy(real_data, MULTIDIM_ARRAY(dest), sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyHostToDevice);
    hipStreamSynchronize(0);
    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */

    /* https://docs.nvidia.com/cuda/cufft/index.html 3.9.3 */

    hipfftExecC2R(planIn, comp_data, real_data);

    hipMemcpy(MULTIDIM_ARRAY(dest),real_data, sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipFree(comp_data);
    hipFree(real_data);
#endif
#ifdef CPU
    FloatPlan p(dest, src2);
    fftw_complex* in = (fftw_complex*) MULTIDIM_ARRAY(src2);
    
    fftw_execute_dft_c2r(plan.getBackward(), in, MULTIDIM_ARRAY(dest));
#endif
}

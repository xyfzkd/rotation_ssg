#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""

#define GPU

#define pi 3.1415926535
#define LENGTH 100000 //signal sampling points

#define DATASIZE 8
#define BATCH 2
/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
#ifdef TEST
    // --- Host side input data allocation and initialization
    hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
    for (int i=0; i<BATCH; i++)
        for (int j=0; j<DATASIZE; j++) hostInputData[i*DATASIZE + j] = (hipfftReal)(i + 1);

    // --- Device side input data allocation and initialization
    hipfftReal *deviceInputData; gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));
    hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

    // --- Host side output data allocation
    hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData; gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;                           // --- 1D FFTs
    int n[] = { DATASIZE };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = BATCH;                      // --- Number of batched executions
    hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);

    //hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
    hipfftExecR2C(handle,  deviceInputData, deviceOutputData);

    // --- Device->Host copy of the results
    gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    for (int i=0; i<BATCH; i++)
        for (int j=0; j<(DATASIZE / 2 + 1); j++)
            printf("%i %i %f %fn", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

    hipfftDestroy(handle);
    gpuErrchk(hipFree(deviceOutputData));
    gpuErrchk(hipFree(deviceInputData));

}
#endif
#ifdef GPU
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);
    /* https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1 */
    hipfftHandle plan;
    hipfftComplex *host_data;
    hipfftReal *real_data;

//    if (hipGetLastError() != hipSuccess){
//        fprintf(stderr, "Cuda error: Failed to allocate\n");
//        return;
//    }

//    hipMalloc((void**)&real_data, sizeof(hipfftComplex)*N[0]*N[1]);
    hipMalloc((void**)&comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));


    hipMemcpy(comp_data, (hipfftComplex*) MULTIDIM_ARRAY(src2), sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);
    printf("nihoa\n");
//    hipDeviceSynchronize();//wait to be done
//    hipMemcpy(real_data, (hipfftReal*) MULTIDIM_ARRAY(dest), sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyHostToDevice);

    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */

    /* https://docs.nvidia.com/cuda/cufft/index.html 3.9.3 */

    hipfftExecC2R(planIn, comp_data, comp_data);

    hipMemcpy(MULTIDIM_ARRAY(dest),comp_data, sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipFree(comp_data);
    hipFree(real_data);
#endif
}

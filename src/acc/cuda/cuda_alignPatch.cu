#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""

#define GPU

#define pi 3.1415926535
#define LENGTH 100000 //signal sampling points

#define DATASIZE 8
#define BATCH 2

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
#ifdef TEST
    // --- Host side input data allocation and initialization
    hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
    for (int i=0; i<BATCH; i++)
        for (int j=0; j<DATASIZE; j++) hostInputData[i*DATASIZE + j] = (hipfftReal)(i + 1);

    // --- Device side input data allocation and initialization
    hipfftReal *deviceInputData; gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));
    hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

    // --- Host side output data allocation
    hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData; gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;                           // --- 1D FFTs
    int n[] = { DATASIZE };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = BATCH;                      // --- Number of batched executions
    hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);

    //hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
    hipfftExecR2C(handle,  deviceInputData, deviceOutputData);

    // --- Device->Host copy of the results
    gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    for (int i=0; i<BATCH; i++)
        for (int j=0; j<(DATASIZE / 2 + 1); j++)
            printf("%i %i %f %fn", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

    hipfftDestroy(handle);
    gpuErrchk(hipFree(deviceOutputData));
    gpuErrchk(hipFree(deviceInputData));
#endif
#ifdef GPU
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);
    /* https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1 */

    hipfftComplex *host_comp_data, *device_comp_data;
    hipfftReal    *host_real_data, *device_real_data;

//    if (hipGetLastError() != hipSuccess){
//        fprintf(stderr, "Cuda error: Failed to allocate\n");
//        return;
//    }
    host_comp_data = (hipfftComplex*) MULTIDIM_ARRAY(src2);
    host_real_data = MULTIDIM_ARRAY(dest);

    gpuErrchk(hipMalloc((void**)&device_real_data, sizeof(hipfftReal)*N[0]*N[1]));
    gpuErrchk(hipMalloc((void**)&device_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1)));


    hipMemcpy(device_comp_data, host_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);
    printf("nihoa\n");

    hipfftHandle planIn;


    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */

    /* https://docs.nvidia.com/cuda/cufft/index.html 3.9.3 */

    hipfftExecC2R(planIn, device_comp_data, device_real_data);

    hipMemcpy(host_real_data, device_real_data, sizeof(hipfftReal)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipfftDestroy(planIn);
    gpuErrchk(hipFree(device_comp_data));
    gpuErrchk(hipFree(device_real_data));
#endif
}

#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);

    /* https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1 */
    hipfftHandle planIn;
    hipfftComplex *comp_data;
    hipfftReal *real_data;

//    if (hipGetLastError() != hipSuccess){
//        fprintf(stderr, "Cuda error: Failed to allocate\n");
//        return;
//    }

    hipMalloc((void**)&real_data, sizeof(hipfftComplex)*N[0]*N[1]);
    hipMalloc((void**)&comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));


    hipMemcpy(comp_data, (hipfftComplex*) MULTIDIM_ARRAY(src2), sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);
    hipMemcpy(real_data, MULTIDIM_ARRAY(dest), sizeof(hipfftComplex)*N[0]*N[1], hipMemcpyHostToDevice);

    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */

    /* https://docs.nvidia.com/cuda/cufft/index.html 3.9.3 */

    if (hipfftExecC2R(planIn, comp_data, real_data) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
        return;
    }
    hipFree(comp_data);
    hipFree(real_data);
}
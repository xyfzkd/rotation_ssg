#include <hipfft/hipfftw.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
//#include "src/acc/cuda/cuda_alignPatch.h"
#include "src/acc/acc_alignPatch.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include ""
#include "time.h"



/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void CuFFT::inverseFourierTransform(
        MultidimArray<fComplex>& src,
        MultidimArray<float>& dest)
{
    /* http://www.orangeowlsolutions.com/archives/1173
     * https://docs.nvidia.com/cuda/cufft/index.html#cufftdoublecomplex 4.2.1
     * https://docs.nvidia.com/cuda/cufft/index.html 3.9.3
     * */
    if (!areSizesCompatible(dest, src))
    {
        resizeRealToMatch(dest, src);
    }

    MultidimArray<fComplex> src2 = src;

    std::vector<int> N(0);
    if (dest.zdim > 1) N.push_back(dest.zdim);
    if (dest.ydim > 1) N.push_back(dest.ydim);
    N.push_back(dest.xdim);

    float elapsedTime = 0;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipfftComplex *host_comp_data, *device_comp_data;
    hipfftReal    *host_real_data, *device_real_data;

    /* https://stackoverflow.com/questions/16511526/cufft-and-fftw-data-structures-are-cufftcomplex-and-fftwf-complex-interchangabl
     * Are hipfftComplex and fftwf_complex interchangable? yes!
     */
    host_comp_data = (hipfftComplex*) MULTIDIM_ARRAY(src2);
    host_real_data = MULTIDIM_ARRAY(dest);

    gpuErrchk(hipMalloc((void**)&device_real_data, sizeof(hipfftReal)*N[0]*N[1]));
    gpuErrchk(hipMalloc((void**)&device_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1)));


    hipMemcpy(device_comp_data, host_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1), hipMemcpyHostToDevice);

    hipfftHandle planIn;


    /* Create a 2D FFT plan. */
    hipfftPlan2d(&planIn,  N[0], N[1], HIPFFT_C2R);


    hipfftExecC2R(planIn, device_comp_data, device_real_data);

    hipMemcpy(host_real_data, device_real_data, sizeof(hipfftReal)*N[0]*N[1], hipMemcpyDeviceToHost);

    hipfftDestroy(planIn);
    gpuErrchk(hipFree(device_comp_data));
    gpuErrchk(hipFree(device_real_data));

    //GET CALCULATION TIME
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("CUFFT Calculation COMPLETED IN : % 5.3f ms \n",elapsedTime);
}
